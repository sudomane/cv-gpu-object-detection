#include "hip/hip_runtime.h"
#include <pipeline.hpp>

#include <err.h>
#include <cmath>
#include <GPU_ops.cuh>
#include <opencv2/opencv.hpp>

template<typename T>
static inline T* _cudaMalloc(size_t n)
{
    T* d_out;

    int rc = hipMalloc(&d_out, sizeof(T) * n);
    if (rc)
      errx(1, "Failed to allocate d_out.");

    return d_out;
}

static inline unsigned char* _toDevice(const unsigned char* h_src, int width, int height, int n_channels = 1)
{
    unsigned char* d_dst = _cudaMalloc<unsigned char>(width * height * n_channels);

    int rc = hipMemcpy(d_dst, h_src, sizeof(unsigned char) * width * height * n_channels, hipMemcpyHostToDevice);
    if (rc)
        errx(1, "Failed to copy host memory to device buffer.");

    return d_dst;
}

static inline void _saveImage(const unsigned char* d_image_data, const t_point& dim, const std::string& filename)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    unsigned char* h_image_data = new unsigned char[width * height];

    int rc = hipMemcpy(h_image_data, d_image_data, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost);

    if (rc)
        errx(1, "Failed to copy host memory to device buffer while saving image.");

    cv::Mat image(width, height, CV_8UC1);
    memcpy(image.data, h_image_data, width * height * sizeof(unsigned char));

    cv::imwrite(filename, image);

    delete[] h_image_data;
}

static inline unsigned char* _initRef(unsigned char* h_ref_image, const t_point& dim)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    int block_size = 256;
    int num_blocks = (width * height + block_size - 1) / block_size;

    unsigned char* h_ref_gray  = new unsigned char[width * height];

    unsigned char* d_ref_gray  = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_ref_image = _toDevice(h_ref_image, width, height, 3);

    GPU::grayscale<<<num_blocks, block_size>>>(d_ref_gray, d_ref_image, width, height);

    delete[] h_ref_gray;

    return d_ref_gray;
}

static inline float* _generateDeviceKernel(int kernel_size, float sigma)
{
    float  sum = 0;
    float* h_kernel = new float[kernel_size * kernel_size];
    float* d_kernel = _cudaMalloc<float>(kernel_size * kernel_size);

    for (int i = 0; i < kernel_size; i++)
    {
        for (int j = 0; j < kernel_size; j++)
        {
            float x = i - kernel_size / 2;
            float y = j - kernel_size / 2;

            float val = std::exp2f(-(x*x + y*y) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
            h_kernel[j * kernel_size + i] = val;

            sum += val;
        }
    }

    for (int i = 0; i < kernel_size; i++)
    {
        for (int j = 0; j < kernel_size; j++)
        {
            h_kernel[j * kernel_size + i] /= sum;
        }
    }

    hipMemcpy(d_kernel, h_kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);

    delete[] h_kernel;

    return d_kernel;
}

void GPU::runPipeline(std::vector<std::pair<std::string, unsigned char*>>& images,
                     const std::pair<int, int> &dim)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    int sigma         = 10;
    int kernel_size   = 21;
    int kernel_offset = std::floor(kernel_size / 2);
    float* d_kernel   = _generateDeviceKernel(kernel_size, sigma);

    int block_size = 256;
    int num_blocks = (width * height + block_size - 1) / block_size;

    unsigned char* d_ref     = _initRef(std::get<1>(images[0]), dim);
    unsigned char* d_buffer  = _cudaMalloc<unsigned char>(width * height * sizeof(unsigned char));
    unsigned char* d_buffer_ = _cudaMalloc<unsigned char>(width * height * sizeof(unsigned char));

    for (int i = 1; i < images.size(); i++)
    {
        const std::string filename = std::get<0>(images[i]);
        unsigned char* h_image     = std::get<1>(images[i]);
        unsigned char* d_image     = _toDevice(h_image, width, height, 3);

        GPU::grayscale <<<num_blocks, block_size>>>(d_buffer, d_image, width, height);
        GPU::difference<<<num_blocks, block_size>>>(d_buffer, d_ref, width, height);
        GPU::gaussian  <<<num_blocks, block_size>>>(d_buffer_, d_buffer, d_kernel,
                                                    width, height, kernel_size,
                                                    sigma, kernel_offset);

        _saveImage(d_buffer_, dim, "out" + std::to_string(i) + ".png");

        std::cout << "[GPU] : " << i << "/" << images.size()-1 << std::endl;

        hipFree(d_image);
    }

    hipFree(d_buffer);
    hipFree(d_buffer_);
    hipFree(d_ref);
    hipFree(d_kernel);
}