#include <pipeline.hpp>

#include <cmath>
#include <GPU_ops.cuh>
#include <opencv2/opencv.hpp>

static inline void _saveImage(const unsigned char* d_image_data, int width, int height, const std::string& filename)
{
    unsigned char* h_image_data = new unsigned char[width * height];

    int rc = hipMemcpy(h_image_data, d_image_data, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost);

    if (rc)
        errx(1, "Failed to copy host memory to device buffer while saving image.");

    cv::Mat image(width, height, CV_8UC1);
    memcpy(image.data, h_image_data, width * height * sizeof(unsigned char));

    cv::imwrite(filename, image);

    delete[] h_image_data;
}

static inline unsigned char* _initRef(unsigned char* h_ref_image, int width, int height)
{
    int block_size = 256;
    int num_blocks = (width * height + block_size - 1) / block_size;

    unsigned char* h_ref_gray  = new unsigned char[width * height];

    unsigned char* d_ref_gray  = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_ref_image = _toDevice<unsigned char>(h_ref_image, width, height, 3);

    GPU::grayscale<<<num_blocks, block_size>>>(d_ref_gray, d_ref_image, width, height);

    delete[] h_ref_gray;

    return d_ref_gray;
}

void GPU::runPipeline(std::vector<std::pair<std::string, unsigned char*>>& images,
                     int width, int height, const json& config, const std::string& bbox_output)
{
    json bbox_JSON_data;

    int bin_thresh     = config["threshold"];
    int sigma          = config["sigma"];
    int kernel_size    = config["kernel_size"];
    int opening_size   = config["opening_size"];
    int closing_size   = config["closing_size"];

    float* d_kernel             = _generateDeviceKernel(kernel_size, sigma);
    int*   d_CC_labels          = _cudaMalloc<int>(width * height);
    unsigned char* d_ref        = _initRef(std::get<1>(images[0]), width, height);
    unsigned char* d_buffer     = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_buffer_tmp = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_buffer_alt = _cudaMalloc<unsigned char>(width * height); // Additional temporary buffer

    GPU::HostWrapper wrapper(width, height, kernel_size, opening_size, closing_size, bin_thresh);

    for (int i = 1; i < images.size(); i++)
    {
        const std::string filename = std::get<0>(images[i]);
        unsigned char* h_image     = std::get<1>(images[i]);
        unsigned char* d_image     = _toDevice<unsigned char>(h_image, width, height, 3);

        wrapper.grayscale  (d_buffer, d_image);
        wrapper.difference (d_buffer, d_ref);
        wrapper.gaussian   (d_buffer_tmp, d_buffer, d_kernel);
        wrapper.morphology (d_buffer, d_buffer_tmp, d_buffer_alt);
        wrapper.binary     (d_buffer);
        wrapper.initLabelCC(d_CC_labels);
        wrapper.components (d_CC_labels);

        //_saveImage(d_buffer, width, height, "out"+std::to_string(i)+".png");

        std::cout << "Processed frame " << i << " of " << images.size()-1 << std::endl;

        hipFree(d_image);

        _addToJSON(bbox_JSON_data, filename, {});
    }

    _exportJSON(bbox_JSON_data, bbox_output);

    hipFree(d_ref);
    hipFree(d_kernel);

    hipFree(d_buffer);
    hipFree(d_buffer_tmp);
    hipFree(d_buffer_alt);
    hipFree(d_CC_labels);
}
