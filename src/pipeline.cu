#include "hip/hip_runtime.h"
#include <pipeline.hpp>

#include <cmath>
#include <GPU_ops.cuh>
#include <opencv2/opencv.hpp>

static inline void _saveImage(const unsigned char* d_image_data, const t_point& dim, const std::string& filename)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    unsigned char* h_image_data = new unsigned char[width * height];

    int rc = hipMemcpy(h_image_data, d_image_data, sizeof(unsigned char) * width * height, hipMemcpyDeviceToHost);

    if (rc)
        errx(1, "Failed to copy host memory to device buffer while saving image.");

    cv::Mat image(width, height, CV_8UC1);
    memcpy(image.data, h_image_data, width * height * sizeof(unsigned char));

    cv::imwrite(filename, image);

    delete[] h_image_data;
}

static inline unsigned char* _initRef(unsigned char* h_ref_image, const t_point& dim)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    int block_size = 256;
    int num_blocks = (width * height + block_size - 1) / block_size;

    unsigned char* h_ref_gray  = new unsigned char[width * height];

    unsigned char* d_ref_gray  = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_ref_image = _toDevice<unsigned char>(h_ref_image, width, height, 3);

    GPU::grayscale<<<num_blocks, block_size>>>(d_ref_gray, d_ref_image, width, height);

    delete[] h_ref_gray;

    return d_ref_gray;
}

void GPU::runPipeline(std::vector<std::pair<std::string, unsigned char*>>& images,
                     const std::pair<int, int> &dim)
{
    int width  = std::get<0>(dim);
    int height = std::get<1>(dim);

    int threshold     = 60; // TODO: Implement adaptative thresholding
    int sigma         = 10;
    int kernel_size   = 21;
    int opening_size  = 21;
    int closing_size  = 21;
    int kernel_offset = std::floor(kernel_size / 2);

    int block_size = 256;
    int num_blocks = (width * height + block_size - 1) / block_size;

    float* d_kernel             = _generateDeviceKernel(kernel_size, sigma);
    unsigned char* d_ref        = _initRef(std::get<1>(images[0]), dim);
    unsigned char* d_buffer     = _cudaMalloc<unsigned char>(width * height);
    unsigned char* d_buffer_tmp = _cudaMalloc<unsigned char>(width * height);

    for (int i = 1; i < images.size(); i++)
    {
        const std::string filename = std::get<0>(images[i]);
        unsigned char* h_image     = std::get<1>(images[i]);
        unsigned char* d_image     = _toDevice<unsigned char>(h_image, width, height, 3);

        GPU::grayscale <<<num_blocks, block_size>>>(d_buffer, d_image, width, height);
        GPU::difference<<<num_blocks, block_size>>>(d_buffer, d_ref, width, height);
        GPU::gaussian  <<<num_blocks, block_size>>>(d_buffer_tmp, d_buffer, d_kernel, width, height, kernel_size, kernel_offset);
        GPU::morphology<<<num_blocks, block_size>>>(d_buffer, d_buffer_tmp, width, height, opening_size, closing_size, kernel_offset);
        GPU::binary    <<<num_blocks, block_size>>>(d_buffer_tmp, threshold, width, height);

        _saveImage(d_buffer_tmp, dim, "out" + std::to_string(i) + ".png");

        std::cout << "[GPU] : " << i << "/" << images.size()-1 << std::endl;

        hipFree(d_image);
    }

    hipFree(d_buffer);
    hipFree(d_buffer_tmp);
    hipFree(d_ref);
    hipFree(d_kernel);
}
