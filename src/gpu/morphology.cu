#include "hip/hip_runtime.h"
#include <GPU_ops.cuh>

__device__ void _kernelFunc(unsigned char* d_dst, const unsigned char* d_src, int dim, int width, int height, int kernel_size, int offset, bool is_dilation)
{
    int x = dim % width;
    int y = dim / width;

    int ii_start, ii_end;
    int jj_start, jj_end;

    _getDeviceBoundaries(x, kernel_size, width,  offset, ii_start, ii_end);
    _getDeviceBoundaries(y, kernel_size, height, offset, jj_start, jj_end);

    unsigned char tmp = is_dilation ? 255 : 0;

    for (int ii = ii_start; ii < ii_end; ii++)
    {
        for (int jj = jj_start; jj < jj_end; jj++)
        {
            unsigned char val = d_src[ii * height + jj];

            if (is_dilation)
            {
                if (val < tmp)
                    tmp = val;
            }

            else
            {
                if (val > tmp)
                    tmp = val;
            }

        }
    }

    d_dst[x * height + y] = tmp;
}

__global__ void GPU::morphology (unsigned char* d_dst, unsigned char* d_src, int width, int height, int opening_size, int closing_size, int offset)
{
    int dim = blockDim.x * blockIdx.x + threadIdx.x;

    int x = dim % width;
    int y = dim / width;

    if (x >= width || y >= height)
        return;

    // FIXME

    // Closing
    {
        _kernelFunc(d_dst, d_src, dim, width, height, opening_size, offset, true);  // Dilation
        _kernelFunc(d_src, d_dst, dim, width, height, opening_size, offset, false); // Erosion
    }

    // Opening
    {
        //_kernelFunc(d_dst, d_src, dim, width, height, opening_size, offset, false); // Erosion
        //_kernelFunc(d_src, d_dst, dim, width, height, opening_size, offset, true);  // Dilation
    }
}
